#include <stdio.h>
#include <hip/hip_runtime.h>

struct Index{
    uint3 block, thread;
};

__global__ void prob_idx_3d (Index* id){
    int b_idx = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
    int t_idx = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;
    int b_dim = blockDim.x * blockDim.y * blockDim.z;
    int position = b_idx * b_dim + t_idx;

    id[position].block = blockIdx;
    id[position].thread = threadIdx;
}

int main (void){
    int g_x = 2, g_y = 2, g_z = 2;
    int b_x = 2, b_y = 2, b_z = 2;
    dim3 grid = dim3(g_x, g_y, g_z);
    dim3 block = dim3(b_x, b_y, b_z);
    printf("gridDim = dim3(%d, %d, %d)\n",grid.x, grid.y, grid.z);
    printf("blockDim = dim3(%d, %d, %d)\n",block.x, block.y, block.z);

    int N = grid.x*grid.y*grid.z*block.x*block.y*block.z;
    printf("Total number of threads = %d\n",N);

    size_t size = N*sizeof(Index);

    Index *h = (Index*)malloc(size);
    memset(h, 0, size);

    Index *d;
    hipMalloc((void**) &d, size);
    hipMemcpy(d, h, size, hipMemcpyHostToDevice);

    prob_idx_3d <<< grid, block >>> (d);

    hipError_t r = hipGetLastError();
    printf("prob_idx_3d: %s\n",hipGetErrorString(r));
    if (r != 0) goto end;

    hipMemcpy(h, d, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++){
        printf("h[%d] = block:(%d, %d, %d), thread:(%d, %d, %d)\n",
            i, h[i].block.x, h[i].block.y, h[i].block.z, h[i].thread.x, h[i].thread.y, h[i].thread.z);

    }

    end:;

    hipFree(d);
    delete h;

    return 0;

}