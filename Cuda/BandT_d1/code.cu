#include<stdio.h>
#include<hip/hip_runtime.h>

struct Index{
        int block, thread;
};

__global__ void prob_idx(Index id[]){
        int b_idx = blockIdx.x;       
        int t_idx = threadIdx.x;      
        int b_dim = blockDim.x;       
        int position = b_idx * b_dim + t_idx;            
        
        id[position].block=b_idx;
        id[position].thread=t_idx;
};

int main(){
        Index* d;
        Index  h[100];

        
        hipMalloc((void**) &d, 100*sizeof(Index));

        int threadsPerBlock = 3;
        int blocksPerGrid = 4;
        int N = threadsPerBlock * blocksPerGrid;
        prob_idx<<< blocksPerGrid, threadsPerBlock>>>(d);

        hipMemcpy(h, d, 100*sizeof(Index), hipMemcpyDeviceToHost);

        for(int i=0; i<N; i++){
            printf("h[%d]={block:%d, thread:%d}\n", i,h[i].block,h[i].thread);
        }

        hipFree(d);
        return 0;
}