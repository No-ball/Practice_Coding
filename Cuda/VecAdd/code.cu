#include<stdio.h>
#include<hip/hip_runtime.h>

struct Position{
    uint3 thread;
    uint3 block;
};

__global__ void VecAdd(float *A,float *B, float *C, Position *p,int n){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    p[i].block = blockIdx;
    p[i].thread = threadIdx;
    if (i<n) C[i] = A[i] + B[i];
}

int main (void){
    int N = 3 * 256 + 1;
    size_t size = N * sizeof(int);
    
    Position *p = (Position*)malloc(N * sizeof(Position));
    memset(p, 0, N * sizeof(Position));

    Position *d_P;
    hipMalloc ((void**) &d_P, N * sizeof(Position));
    hipMemcpy(d_P, p, N * sizeof(Position), hipMemcpyHostToDevice);


    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float h_C[1000];

    for (int i=0; i<N; i++){
        h_A[i] = i;
        h_B[i] = (float)i/1000;
    }

    float* d_A;
    float* d_B;
    float* d_C;

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    VecAdd <<< blocksPerGrid, threadsPerBlock >>> (d_A, d_B, d_C, d_P, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    hipMemcpy(p, d_P, N * sizeof(Position), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    for (int i = 0; i<N; i++){
        printf("block:(%d, %d, %d), thread:(%d, %d, %d) -- %f + %f = %f\n", p[i].block.x, p[i].block.y, p[i].block.z, p[i].thread.x, p[i].thread.y, p[i].thread.z, h_A[i] ,h_B[i] ,h_C[i]);
    }
    delete h_A;
    delete h_B;
    delete p;

    return 0;
}