#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<math.h>
#include<hip/hip_runtime.h>

void transpose_host (float *b, float *a, int m, int n){
    for (int y = 0; y < m; y++){
        for (int x = 0; x < n; x ++){
            b[x*m + y] = a[y*n + x];
        }
    }
}

__global__ void transpose_naive_cr(float *b, float *a, int m, int n){
    int x = blockIdx.x*blockDim.x + threadIdx.y;
    int y = blockIdx.y*blockDim.y + threadIdx.x;
    if (y < m && x < n){
        b[x*m + y] = a[y*n + x];
    }
}

__global__ void transpose_shared(float *b, float *a, int m, int n){
    __shared__ float s[256];

    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    if (y < m && x < n){
        int t = threadIdx.y*blockDim.x + threadIdx.x;
        int i = y*n + x;
        s[t] = a[i]; 
    }
    __syncthreads();

    x = blockIdx.x*blockDim.x + threadIdx.y;
    y = blockIdx.y*blockDim.y + threadIdx.x;

    if (y < m && x < n){
        int t = threadIdx.x*blockDim.y + threadIdx.y;
        int o = x*m + y;
        b[o] = s[t];
    }
}

double rd(float *a, float *b, int size){
    double s = 0, d = 0;
    for (int k = 0; k < size; k++){
        double w = a[k] - b[k];
        s += a[k]*a[k];
        d += w*w;
    }
    return sqrt(d/s);
}

timer_t timer[10];
void set_timer(int k = 0){
    timer[k] = clock();
}
double get_timer[10]{
    return (double)(clock() - timer[k])/CLOCKS_PER_SEC;
}
void test(int m, int n, int loop = 100; int loop_host = 10){
    size_t size = m * n;
    printf("matrix size: %d x %d\n", m, n);

    float *a = new float[size];
    float *b = new float[size];
    float *c = new float[size];

    for (int i = 0; i < size; i ++){
        a[i] = (float)rand()*2/RAND_MAX-1;
        b[i] = 0;
    }

    float *ga, *gb;
    hipMalloc((void**) &ga, size * sizeof(float));
    hipMalloc((void**) &gb, size * sizeof(float));
    hipMemcpy(ga, a, size*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(gb, b, size*sizeof(float), hipMemcpyDeviceToHost);

    dim3 grid(n/16 + 1, m/16 + 1, 1);
    dim3 block(16,16,1);

    set_timer();
    for (int i = 0; i < loop_host; i++){
        transpose_host(b,a,m,n);
    }
    double t0 = get_timer()/loop_host;
    printf("host\ttime: %g ms\n", t0*1000);

    hipMemset(gb, 0, size*sizeof(float));
    transpose_naive_cr <<< grid, block >>> (gb, ga, m, n);
    hipMemcpy(c, gb, size * sizeof(float), hipMemcpyDeviceToHost);

    set_timer();
    hipDeviceSynchronize();
    for (int i = 0; i < loop; i ++){
        transpose_naive_cr <<< grid, block >>> (bg, ga, m, n);
    }
    hipDeviceSynchronize();
    double t1 = get_timer()/loop;
    printf("naive(r)\ttime: %g ms (%dx)\terror: %g\n", t1*1000,(int)(t0/t1),rd(b, c, size));

    
}