#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

//----------------------------------------------
// 排序 N 個 float 元素 (N=1~1025)
// 使用到的記憶體大小為 SIZE 個 bytes
// 只使用單一區塊
// 區塊大小為 N/2
//----------------------------------------------
#define N      1024
#define SIZE   (N*sizeof(float))

#define GRID   1
#define BLOCK  (N/2)

#define testLoop 1000  //測試效能時的 loop 數

//----------------------------------------------
// 交換函式 (host 和 kernel 都可以使用) 
// 因為加了 __host__ 和 __device__ 兩個標籤
//----------------------------------------------
inline __host__ __device__ void swap(float& a, float& b){
        float c=a;
        a=b;
        b=c;
}

//----------------------------------------------
// 泡泡的 kernel (由小到大排列 N 個元素 a->r)
//----------------------------------------------
__global__ void bubble(float *r, float *a){
        //*** blockDim=N/2 ***
        int j=threadIdx.x;      //j=0,1,2,...blockDim-1 
        int k=2*threadIdx.x;    //k=0,2,4,...2*(blockDim-1) 配對的基底索引

        //配置共享記憶體
        __shared__ float s[N+20];

        //載入資料到共享記憶體
        __syncthreads();   //同步化執行緒, 加速載入速度 (合併讀取 coalesced)
        s[j]=a[j];         //使用全部執行緒一起載入前半段 (0~N/2-1)
        s[j+N/2]=a[j+N/2]; //使用全部執行緒一起載入後半段 (N/2~N-1)
        if(j==0){
                //若 N 為奇數時, 還要多載入一個尾巴, 只使用第 0 個執行緒
                s[N-1]=a[N-1];
        }

        //開始泡泡排序
        for(int loop=0; loop<=N/2; loop++){
                //排列 0 based 配對資料 (0,1) (2,3) (4,5) .... 
                __syncthreads();  //同步化確保共享記憶體已寫入
                if(s[k]>s[k+1]){
                        swap(s[k],s[k+1]);
                }

                //排列 1 based 配對資料 (1,2) (3,4) (5,6) .... 
                __syncthreads();  //同步化確保共享記憶體已寫入
                if(s[k+1]>s[k+2]){
                        if(k<N-2) //若 N 為偶數時, 最後一個執行緒不作用
                        swap(s[k+1],s[k+2]);
                }
        }

        //轉出資料到全域記憶體
        __syncthreads();
        r[j]=s[j];
        r[j+N/2]=s[j+N/2];
        if(j==0){
                r[N-1]=s[N-1];
        }

}


//----------------------------------------------
// 泡泡的 host 函數 
//----------------------------------------------
void bubble_host(float *r, float *a){
        //載入資料
        for(int k=0; k<N; k++){
                r[k]=a[k];
        }

        for(int loop=0; loop<=N/2; loop++){
                //排列 0 based 配對資料
                for(int k=0; k<N-1; k+=2){
                        if(r[k]>r[k+1]){
                                swap(r[k],r[k+1]);
                        }
                }
                //排列 1 based 配對資料
                for(int k=1; k<N-1; k+=2){
                        if(r[k]>r[k+1]){
                                swap(r[k],r[k+1]);
                        }
                }
        }
}


//----------------------------------------------
// 主程式
//----------------------------------------------
int main(){
        //配置 host 記憶體
        float *a=(float*)malloc(SIZE);
        float *b=(float*)malloc(SIZE);
        float *c=(float*)malloc(SIZE);


        //初始化
        for(int k=0; k<N; k++){
                a[k]=k;
                c[k]=0;
        }

        //對陣列 a 洗牌
        srand(time(0));
        for(int k=0; k<2*N; k++){
                int i=rand()%N;
                int j=rand()%N;
                swap(a[i],a[j]);
        }

        //配置 device 記憶體
        float  *ga, *gc;
        hipMalloc((void**)&ga, SIZE);
        hipMalloc((void**)&gc, SIZE);

        //載入 (順便載入 c 來清空裝置記憶體內容)
        hipMemcpy(ga, a, SIZE, hipMemcpyHostToDevice);
        hipMemcpy(gc, c, SIZE, hipMemcpyHostToDevice);


        //測試 kernel 效能
        double t0=(double)clock()/CLOCKS_PER_SEC;
                for(int k=0; k<testLoop; k++){
                        //呼叫 kernel (此為單一 block 的版本)
                        bubble<<<1,BLOCK>>>(gc,ga);

                        //同步化執行緒, 避免還沒做完, 量到不正確的時間
                        hipDeviceSynchronize();
                }
        t0=((double)clock()/CLOCKS_PER_SEC-t0)/testLoop;

        //測試 host 效能
        double t1=(double)clock()/CLOCKS_PER_SEC;
                for(int k=0; k<testLoop; k++){
                        bubble_host(b,a);
                }
        t1=((double)clock()/CLOCKS_PER_SEC-t1)/testLoop;

        //顯示計算時間, 並比較
        printf("time[gpu]: %g ms\n",t0*1000);
        printf("time[host]: %g ms\n",t1*1000);
        printf("ratio: %g x\n",t1/t0);

        //讀出 device 資料
        hipMemcpy(c, gc, SIZE, hipMemcpyDeviceToHost);


        //測試 device 結果的正確性
        printf("------------------------\n");
        bool flag=true;
        for(int k=0; k<N; k++){
                if(c[k]!=k){
                        flag=false;
                        break;
                }
        }
        printf("test[gpu]: %s\n",flag?"pass":"fail");

        //測試 host 結果的正確性
        flag=true;
        for(int k=0; k<N; k++){
                if(b[k]!=k){
                        flag=false;
                        break;
                }
        }
        printf("test[host]: %s\n",flag?"pass":"fail");

        //釋放記憶體
        hipFree(ga);
        hipFree(gc);
        free(a);
        free(b);
        free(c);


        return 0;
}
