#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void test (int input, double *num){
    int idx = threadIdx.x;
    num[idx] =input-num[idx]*num[idx];
}
void run (int input, double *ans, int digit, double timer){
    
    if (digit <= 0) return;

    double *r;
    double *num = new double[100];
    for (int i=0; i<100; i++){
        num[i] = *ans + timer*i;
    }

    hipMalloc((void**) &r, 100*sizeof(double));
    hipMemcpy(r, num, 100*sizeof(double), hipMemcpyHostToDevice);

    test <<< 1,100 >>> (input,r);

    hipMemcpy(num, r, 100*sizeof(double), hipMemcpyDeviceToHost);
    hipFree(r);

    for (int i=0; i<100; i++){
        if (num[i] < 0){
            *ans += timer*(i-1);
            break; 
        }else if (i == 99) {
            *ans += timer*99;
        }
    }

    delete num;
    run(input,ans,digit - 1,timer/100);
}
int find_timer(int input){
    int ans = 1, i = 100;
    while(input /= i) ans *= 100;
    //printf("%d",ans);
    return ans;
}
int main (void){
    //char trash[100+5];
    int digit,input;
    //scanf("%s %s",&trash,&trash);
    scanf("%d %d",&digit,&input);
    int timer = find_timer(input);
    double *ans = new double;
    run(input, ans, digit, timer);
    printf("%.10f\n",*ans);
    delete ans;
    return 0;
}