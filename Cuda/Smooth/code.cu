#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<time.h>

#define Block 512

void c_smooth (float *a, float *b, int N){
    for (int i = 1; i < N; i++){
        b[i] = (a[i-1] + 2*a[i] + a[i+1]) * 0.25;
    }
    b[0] = (2*a[0] + a[1]) * 0.25;
    b[N-1] = (a[N-2] + 2*a[N-1]) * 0.25;
}

__global__ void g_smooth (float *a,float *b,int N){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i == 0) b[i] = (2*a[0] + a[1]) * 0.25;
    else if (i == N-1) b[i] = (a[N-2] + 2*a[N-1]) * 0.25;
    else b[i] = (a[i-1] + 2*a[i] + a[i+1]) * 0.25;
}

__global__ void s_smooth (float *a, float *b,int N){
    int base = blockIdx.x * blockDim.x;
    int t_idx = threadIdx.x;

    __shared__ float s[Block+2];
    
    if (base+t_idx < N) s[t_idx+1] = a[base + t_idx];

    if (t_idx == 0){
        if (base == 0) s[0] = 0;
        else s[0] = a[base-1];
    }
    if (t_idx == 32){
        if (base + Block >= N) s[N-base+1] = 0;
        else s[Block+1] = a[base+Block];
    }

    __syncthreads();

    if (base+t_idx < N) b[base+t_idx] = (s[t_idx] + 2*s[t_idx+1] + s[t_idx+2]) * 0.25;
}

int main (void){
    int N = 10*1000*1000;
    int loop = 130;
    
    float* h_arr = new float[N];
    float* g_arr = new float[N];
    float* s_arr = new float[N];
    float* c_arr = new float[N];
    float* d_arr1;
    float* d_arr2;
    size_t size = N * sizeof(float);

    for (int i = 0; i < N; i++){
        h_arr[i] = (float)rand()/RAND_MAX;
    }
    hipMalloc((void**) &d_arr1, size);
    hipMalloc((void**) &d_arr2, size);
    hipMemcpy(d_arr1, h_arr, size, hipMemcpyHostToDevice);
    hipMemcpy(d_arr2, h_arr, size, hipMemcpyHostToDevice);
    int threadsPerBlocks = 512;
    int blocksPerGrid = N/512 + 1;

    double c_time = (double)clock()/CLOCKS_PER_SEC;
    for (int i = 0; i < loop; i++){
        c_smooth(h_arr, c_arr, N);
    }
    c_time = (double)clock()/CLOCKS_PER_SEC;

    double g_time = (double)clock()/CLOCKS_PER_SEC;
    hipDeviceSynchronize();
    for (int i = 0; i < loop; i++){
        g_smooth <<< blocksPerGrid, threadsPerBlocks >>> (d_arr1,d_arr2,N);
    }
    hipDeviceSynchronize();
    g_time = ((double)clock()/CLOCKS_PER_SEC - g_time);
    hipMemcpy(g_arr, d_arr2, size, hipMemcpyDeviceToHost);

    double s_time = (double)clock()/CLOCKS_PER_SEC;
    hipDeviceSynchronize();
    for (int i = 0; i < loop; i++){
        s_smooth <<< blocksPerGrid, threadsPerBlocks >>> (d_arr1, d_arr2, N);
    }
    hipDeviceSynchronize();
    s_time = ((double)clock()/CLOCKS_PER_SEC - s_time);
    hipMemcpy(s_arr, d_arr2, size, hipMemcpyDeviceToHost);

    float sum_c, sum_s, sum_g;

    for (int i = 0; i < N; i++){
        double diff_sc = s_arr[i] - c_arr[i];
        double diff_gc = g_arr[i] - c_arr[i]; 
        sum_c += c_arr[i] * c_arr[i];
        sum_s += diff_sc * diff_sc;
        sum_g += diff_gc * diff_gc;         
    }
    int num = 2;
    printf("time: %g s\n",(double)clock()/CLOCKS_PER_SEC);
    printf("vector size: %d\n", N);
    printf("loop time: %d\n", loop);
    printf("\nglobal run time: %g ms\n", g_time*1000);
    printf("shared run time: %g ms\n", s_time*1000);
    printf("cpu run time: %g ms\n", c_time*1000);
    printf("c_arr [%d]: %g\n",num ,c_arr[num]);
    printf("g_arr [%d]: %g\n",num ,g_arr[num]);
    printf("s_arr [%d]: %g\n",num ,s_arr[num]);
    printf("Diff(g/c): %g\n",sqrt(sum_g/sum_c));
    printf("Diff(s/c): %g\n",sqrt(sum_s/sum_c));

    hipFree(d_arr1);
    hipFree(d_arr2);
    delete [] h_arr;
    delete [] g_arr;
    delete [] s_arr;
    delete [] c_arr;

    return 0;
}
